#include "hip/hip_runtime.h"
// #include <pycuda-complex.hpp>
#include <surface_functions.h>
#include <stdint.h>
#include <hip/hip_runtime.h>
// #define pi 3.14159265f

texture< float, hipTextureType3D, hipReadModeElementType> tex_tempIn;
surface< void, cudaSurfaceType3D> surf_tempOut;

///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
///////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
__device__ float heatCore( int nWidth, int nHeight, float nDepth, float t, float xMin, float yMin, float zMin, 
				 float dx, float dy, float dz, int t_i, int t_j, int t_k ){
 
  float center, right, left, up, down, top, bottom, result, laplacian;
  center = tex3D(tex_tempIn, (float)t_j, (float)t_i, (float)t_k);
  up =     tex3D(tex_tempIn, (float)t_j, (float)t_i+1, (float)t_k);
  down =   tex3D(tex_tempIn, (float)t_j, (float)t_i-1, (float)t_k);
  right =  tex3D(tex_tempIn, (float)t_j+1, (float)t_i, (float)t_k);
  left =   tex3D(tex_tempIn, (float)t_j-1, (float)t_i, (float)t_k);
  top =    tex3D(tex_tempIn, (float)t_j, (float)t_i, (float)t_k+1);
  bottom = tex3D(tex_tempIn, (float)t_j, (float)t_i, (float)t_k-1);

  //Boundry Conditions
  if (t_j==0 ) left = 0.85f;
  if (t_j==nWidth-1) right = 0.f;
  if (t_i==0) down = 0.85f;
  if (t_i==nHeight-1) up = 0.f;
  if (t_k==0) bottom = 0.f;
  if (t_k==nDepth-1) top = 0.f;

  float dxInv = 1.0f/dx;
  float dyInv = 1.0f/dy;
  float dzInv = 1.0f/dz;

  laplacian = (up + down - 2.f*center )*dyInv*dyInv + (right + left - 2.f*center )*dxInv*dxInv + (top + bottom - 2.f*center )*dzInv*dzInv;
  result = laplacian;

  return result;
}

__global__ void euler_kernel_shared( int nWidth, int nHeight, int nDepth, float slopeCoef, float weight, 
				      float xMin, float yMin, float zMin, float dx, float dy, float dz, float t, float dt, 
				      float *tempFirst,
				      float *inputTemp, float *outputTemp,
				      float *tempRunge,
				      int lastRK4Step){
  int t_j = blockIdx.x*blockDim.x + threadIdx.x;
  int t_i = blockIdx.y*blockDim.y + threadIdx.y;
  int t_k = blockIdx.z*blockDim.z + threadIdx.z;
  int tid = t_j + t_i*blockDim.x*gridDim.x + t_k*blockDim.x*gridDim.x*blockDim.y*gridDim.y;

  float dxInv = 1.0f/dx;
  float dyInv = 1.0f/dy;
  float dzInv = 1.0f/dz;
  
  //copy data to shared memory
  int tid2 = tid;
  const int nNeigh = 2;  //Number of neighbors for spatial derivatives
  int t_x = threadIdx.x + nNeigh;
  int t_y = threadIdx.y + nNeigh;
  int t_z = threadIdx.z + nNeigh;
  float val;
  __shared__ float shrd_temp[ %(BLOCK_WIDTH)s + 2*nNeigh ][ %(BLOCK_HEIGHT)s + 2*nNeigh ][ %(BLOCK_DEPTH)s + 2*nNeigh ];
  shrd_temp[t_x][t_y][t_z] = inputTemp[tid2];
  //fill x halo
  if ( t_x<2*nNeigh ){
    tid2 = (t_j-nNeigh) + t_i*blockDim.x*gridDim.x + t_k*blockDim.x*gridDim.x*blockDim.y*gridDim.y;
    if (blockIdx.x==0) val = 0.5f;
    else val = inputTemp[tid2];
    shrd_temp[t_x-nNeigh][t_y][t_z] = val;
    tid2 = (t_j) + t_i*blockDim.x*gridDim.x + t_k*blockDim.x*gridDim.x*blockDim.y*gridDim.y;
  }
  if ( t_x >= blockDim.x ){
    tid2 = (t_j+nNeigh) + t_i*blockDim.x*gridDim.x + t_k*blockDim.x*gridDim.x*blockDim.y*gridDim.y;
    if (blockIdx.x == gridDim.x-1) val = 0.5f;
    else val = inputTemp[tid2];
    shrd_temp[t_x+nNeigh][t_y][t_z] = val;
    tid2 = (t_j) + t_i*blockDim.x*gridDim.x + t_k*blockDim.x*gridDim.x*blockDim.y*gridDim.y;
  }
  //fill y halo
  if ( t_y<2*nNeigh ){
    tid2 = (t_j) + (t_i-nNeigh)*blockDim.x*gridDim.x + t_k*blockDim.x*gridDim.x*blockDim.y*gridDim.y;
    if (blockIdx.y==0) val = 0.f;
    else val = inputTemp[tid2];
    shrd_temp[t_x][t_y-nNeigh][t_z] = val;
    tid2 = (t_j) + (t_i)*blockDim.x*gridDim.x + t_k*blockDim.x*gridDim.x*blockDim.y*gridDim.y;
  }
  if ( t_y >= blockDim.y ){
    tid2 = (t_j) + (t_i+nNeigh)*blockDim.x*gridDim.x + t_k*blockDim.x*gridDim.x*blockDim.y*gridDim.y;
    if (blockIdx.y == gridDim.y-1) val = 0.f;
    else val = inputTemp[tid2];
    shrd_temp[t_x][t_y+nNeigh][t_z] = val;
    tid2 = (t_j) + (t_i)*blockDim.x*gridDim.x + t_k*blockDim.x*gridDim.x*blockDim.y*gridDim.y;
  }  
  //fill z halo
  if ( t_z<2*nNeigh ){
    tid2 = (t_j) + (t_i)*blockDim.x*gridDim.x + (t_k-nNeigh)*blockDim.x*gridDim.x*blockDim.y*gridDim.y;
    if (blockIdx.z==0) val = 0.f;
    else val = inputTemp[tid2];
    shrd_temp[t_x][t_y][t_z-nNeigh] = val;
    tid2 = (t_j) + (t_i)*blockDim.x*gridDim.x + (t_k)*blockDim.x*gridDim.x*blockDim.y*gridDim.y;
  }
  if ( t_z >= blockDim.z ){
    tid2 = (t_j) + (t_i)*blockDim.x*gridDim.x + (t_k+nNeigh)*blockDim.x*gridDim.x*blockDim.y*gridDim.y;
    if (blockIdx.z == gridDim.z-1) val = 0.f;
    else val = inputTemp[tid2];
    shrd_temp[t_x][t_y][t_z+nNeigh] = val;
    tid2 = (t_j) + (t_i)*blockDim.x*gridDim.x + (t_k)*blockDim.x*gridDim.x*blockDim.y*gridDim.y;
  }    
  __syncthreads(); 
  
  
//   //fill x halo
//   t_x -= 1;
//   t_j -= 1;
//   tid2 = t_j + t_i*blockDim.x*gridDim.x + t_k*blockDim.x*gridDim.x*blockDim.y*gridDim.y;
//   if (t_x < 1){
//     if (blockIdx.x == 0 ) shrd_temp[t_x][t_y][t_z] = 0.5f;
//     else shrd_temp[t_x][t_y][t_z] = inputTemp[tid2];
//   }
//   t_x += 2;
//   t_j += 2;
//   tid2 = t_j + t_i*blockDim.x*gridDim.x + t_k*blockDim.x*gridDim.x*blockDim.y*gridDim.y;
//   if (t_x > blockDim.x ) {
//     if (blockIdx.x == gridDim.x-1 ) shrd_temp[t_x][t_y][t_z] = 0.5f;
//     else shrd_temp[t_x][t_y][t_z] = inputTemp[tid2];
//   }
//   t_x -= 1;
//   t_j -= 1;  
//   //fill y halo
//   t_y -= 1;
//   t_i -= 1;
//   tid2 = t_j + t_i*blockDim.x*gridDim.x + t_k*blockDim.x*gridDim.x*blockDim.y*gridDim.y;
//   if (t_y < 1){
//     if (blockIdx.y == 0 ) shrd_temp[t_x][t_y][t_z] = 0.f;
//     else shrd_temp[t_x][t_y][t_z] = inputTemp[tid2];
//   }
//   t_y += 2;
//   t_i += 2;
//   tid2 = t_j + t_i*blockDim.x*gridDim.x + t_k*blockDim.x*gridDim.x*blockDim.y*gridDim.y;
//   if (t_y > blockDim.y ) {
//     if (blockIdx.y == gridDim.y-1 ) shrd_temp[t_x][t_y][t_z] = 0.f;
//     else shrd_temp[t_x][t_y][t_z] = inputTemp[tid2];
//   }
//   t_y -= 1;
//   t_i -= 1;
//   //fill z halo
//   t_z -= 1;
//   t_k -= 1;
//   tid2 = t_j + t_i*blockDim.x*gridDim.x + t_k*blockDim.x*gridDim.x*blockDim.y*gridDim.y;
//   if (t_z < 1){
//     if (blockIdx.z == 0 ) shrd_temp[t_x][t_y][t_z] = 0.f;
//     else shrd_temp[t_x][t_y][t_z] = inputTemp[tid2];
//   }
//   t_z += 2;
//   t_k += 2;
//   tid2 = t_j + t_i*blockDim.x*gridDim.x + t_k*blockDim.x*gridDim.x*blockDim.y*gridDim.y;
//   if (t_z > blockDim.z ){
//     if (blockIdx.z == gridDim.z-1 ) shrd_temp[t_x][t_y][t_z] = 0.f;
//     else shrd_temp[t_x][t_y][t_z] = inputTemp[tid2];
//   }
//   t_z -= 1;
//   t_k -= 1;
//   __syncthreads(); 


//   float val1, val2;
  float center = shrd_temp[t_x][t_y][t_z];
  float laplacian = 0.f;
  //Add x derivative
  laplacian += ( shrd_temp[t_x-1][t_y][t_z] + shrd_temp[t_x+1][t_y][t_z] - 2.f*center)*dxInv*dxInv;
  //Add y derivative
  laplacian += ( shrd_temp[t_x][t_y-1][t_z] + shrd_temp[t_x][t_y+1][t_z] - 2.f*center)*dyInv*dyInv;
  //Add z derivative
  laplacian += ( shrd_temp[t_x][t_y][t_z-1] + shrd_temp[t_x][t_y][t_z+1] - 2.f*center)*dzInv*dzInv; 
  
  
  
  
  
  
  
  
  
  
  
  
  
  
  
  
//   //copy data to shared memory
//   int tid2;
//   __shared__ float shrd_temp[ %(BLOCK_WIDTH)s + 2 ][ %(BLOCK_HEIGHT)s + 2 ][ %(BLOCK_DEPTH)s + 2 ];
//   int t_x = threadIdx.x;
//   int t_y = threadIdx.y;
//   int t_z = threadIdx.z;
//   t_j -= 1;
//   t_i -= 1;
//   t_k -= 1;
//   tid2 = t_j + t_i*blockDim.x*gridDim.x + t_k*blockDim.x*gridDim.x*blockDim.y*gridDim.y;
//   shrd_temp[t_x][t_y][t_z] = inputTemp[tid2];
//   t_j += 2;
//   t_x += 2;
//   tid2 = t_j + t_i*blockDim.x*gridDim.x + t_k*blockDim.x*gridDim.x*blockDim.y*gridDim.y;
//   if (t_x > nWidth-2) shrd_temp[t_x][t_y][t_z] = inputTemp[tid2];
//   t_i += 2;
//   t_y += 2;
//   tid2 = t_j + t_i*blockDim.x*gridDim.x + t_k*blockDim.x*gridDim.x*blockDim.y*gridDim.y;
//   if ( (t_y > nHeight-2) and (t_x > nWidth-2) ) shrd_temp[t_x][t_y][t_z] = inputTemp[tid2];
//   t_k += 2;
//   t_z += 2;  
//   tid2 = t_j + t_i*blockDim.x*gridDim.x + t_k*blockDim.x*gridDim.x*blockDim.y*gridDim.y;
//   if ( (t_z > nDepth-2) and (t_y > nHeight-2) and (t_x > nWidth-2) ) shrd_temp[t_x][t_y][t_z] = inputTemp[tid2];
//   __syncthreads();
  

   
//   //GLOBAL MEMORY
//   float center = inputTemp[tid];
//   float laplacian = 0.f;
//   int tid_1, tid_2;
//   float val1, val2;
//   //Add x derivative
//   tid_1 = (t_j+1) + t_i*blockDim.x*gridDim.x + t_k*blockDim.x*gridDim.x*blockDim.y*gridDim.y;
//   tid_2 = (t_j-1) + t_i*blockDim.x*gridDim.x + t_k*blockDim.x*gridDim.x*blockDim.y*gridDim.y;
//   val1 = inputTemp[tid_1];
//   val2 = inputTemp[tid_2];
//   if (t_j == 0) val2 = 0.8f;
//   if (t_j == nWidth -1 ) val1 = 0.0f;
//   laplacian += ( val1 + val2 - 2.f*center)*dxInv*dxInv;
//   //Add y derivative
//   tid_1 = t_j + (t_i+1)*blockDim.x*gridDim.x + t_k*blockDim.x*gridDim.x*blockDim.y*gridDim.y;
//   tid_2 = t_j + (t_i-1)*blockDim.x*gridDim.x + t_k*blockDim.x*gridDim.x*blockDim.y*gridDim.y;
//   val1 = inputTemp[tid_1];
//   val2 = inputTemp[tid_2];
//   if (t_i == 0) val2 = 0.8f;
//   if (t_i == nHeight -1 ) val1 = 0.0f;
//   laplacian += ( val1 + val2 - 2.f*center)*dyInv*dyInv;
//   //Add z derivative
//   tid_1 = t_j + t_i*blockDim.x*gridDim.x + (t_k+1)*blockDim.x*gridDim.x*blockDim.y*gridDim.y;
//   tid_2 = t_j + t_i*blockDim.x*gridDim.x + (t_k-1)*blockDim.x*gridDim.x*blockDim.y*gridDim.y;
//   val1 = inputTemp[tid_1];
//   val2 = inputTemp[tid_2];
//   if (t_k == 0) val2 = 0.8f;
//   if (t_k == nDepth -1 ) val1 = 0.0f;
//   laplacian += ( val1 + val2 - 2.f*center)*dzInv*dzInv;
  
  float increment = dt * laplacian;
  

  if (lastRK4Step )
    tempRunge[tid] = tempRunge[tid] + slopeCoef*increment/6.0f;
  else{
    outputTemp[tid] = tempFirst[tid] + weight*increment;
    tempRunge[tid] = tempRunge[tid] + slopeCoef*increment/6.0f;
  }
}
////////////////////////////////////////////////////////////////////////////////
//////////////////////           EULER                //////////////////////////
////////////////////////////////////////////////////////////////////////////////
__global__ void euler_kernel_texture( int nWidth, int nHeight, int nDepth, float slopeCoef, float weight, 
				      float xMin, float yMin, float zMin, float dx, float dy, float dz, float t, float dt, 
				      float *psi1Real_d,
				      float *psiRungeReal,
				      int lastRK4Step){
  int t_j = blockIdx.x*blockDim.x + threadIdx.x;
  int t_i = blockIdx.y*blockDim.y + threadIdx.y;
  int t_k = blockIdx.z*blockDim.z + threadIdx.z;
  int tid = t_j + t_i*blockDim.x*gridDim.x + t_k*blockDim.x*gridDim.x*blockDim.y*gridDim.y;

  float incrementReal;  
  incrementReal = dt * heatCore( nWidth, nHeight, nDepth, t, xMin, yMin, zMin, dx, dy, dz, t_i, t_j, t_k);
//   incrementReal = dt*incrementReal;
 
  float valueReal;
  if (lastRK4Step ){
    valueReal = psiRungeReal[tid] + slopeCoef*incrementReal/6.0f;
    psiRungeReal[tid] = valueReal;
    surf3Dwrite(  valueReal, surf_tempOut,  t_j*sizeof(float), t_i, t_k,  hipBoundaryModeClamp);
  }
  
  else{
    valueReal = psi1Real_d[tid] + weight*incrementReal;
    surf3Dwrite(  valueReal, surf_tempOut,  t_j*sizeof(float), t_i, t_k,  hipBoundaryModeClamp);
    //add to rk4 final value
    psiRungeReal[tid] = psiRungeReal[tid] + slopeCoef*incrementReal/6.0f;
  }
}